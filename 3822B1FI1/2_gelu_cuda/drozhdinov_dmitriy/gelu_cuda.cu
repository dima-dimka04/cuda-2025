#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <>

const float sqrt2pi = 0.797884f;

__global__ void kernel(const float* sample, float* result, size_t elemCount) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < elemCount) {
    const float num = sample[i];
    result[i] = 0.5f * num * (1.0f + tanhf(sqrt2pi * num * (1.0f + 0.044715f * num * num)));
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  const size_t size = input.size();
  std::vector<float> output(size);

  size_t Bytes = size * sizeof(*input.data());

  float* d_input;
  float* d_output;
  hipMalloc(&d_input, Bytes);
  hipMalloc(&d_output, Bytes);

  hipMemcpy(d_input, input.data(), Bytes, hipMemcpyHostToDevice);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
  size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

  hipMemcpy(output.data(), d_output, Bytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
  return output;
}